#include "hip/hip_runtime.h"
#include "poisson2d.hpp"
#include "timer.hpp"
#include <algorithm>
#include <iostream>
#include <stdio.h>

// y = A * x
__global__ void cuda_csr_matvec_product(int N, int *csr_rowoffsets,
                                        int *csr_colindices, double *csr_values,
                                        double *x, double *y)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
  {
    double sum = 0;
    for (int k = csr_rowoffsets[i]; k < csr_rowoffsets[i + 1]; k++)
    {
      sum += csr_values[k] * x[csr_colindices[k]];
    }
    y[i] = sum;
  }
}

// x <- x + alpha * y
__global__ void cuda_vecadd(int N, double *x, double *y, double alpha)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
    x[i] += alpha * y[i];
}

// x <- y + alpha * x
__global__ void cuda_vecadd2(int N, double *x, double *y, double alpha)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
    x[i] = y[i] + alpha * x[i];
}

// result = (x, y)
__global__ void cuda_dot_product(int N, double *x, double *y, double *result)
{
  __shared__ double shared_mem[512];

  double dot = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
  {
    dot += x[i] * y[i];
  }

  shared_mem[threadIdx.x] = dot;
  for (int k = blockDim.x / 2; k > 0; k /= 2)
  {
    __syncthreads();
    if (threadIdx.x < k)
    {
      shared_mem[threadIdx.x] += shared_mem[threadIdx.x + k];
    }
  }

  if (threadIdx.x == 0)
    atomicAdd(result, shared_mem[0]);
}

////////////// CG KERNEL 1: //////////////

__global__ void cuda_cg_1(int N,
                          double alpha,
                          double beta,
                          double *x,
                          double *r,
                          double *p,
                          double *Ap,
                          double *result_rr)
{

  __shared__ double shared_mem[512];
  double dot = 0;

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
  {
    // line 2,3,4: get x, r, p
    x[i] = x[i] + alpha * p[i];
    r[i] = r[i] - alpha * Ap[i];
    p[i] = r[i] + beta * p[i];
    
    // line 6: get dot(r,r)
    dot += r[i] * r[i];
  }

  __syncthreads();
  shared_mem[threadIdx.x] = dot;

  for (int k = blockDim.x / 2; k > 0; k /= 2)
  {
    __syncthreads();
    if (threadIdx.x < k)
    {
      shared_mem[threadIdx.x] += shared_mem[threadIdx.x + k];
    }
  }

  if (threadIdx.x == 0)
  {
    atomicAdd(result_rr, shared_mem[0]);
    // printf("%g", r[0]);
  }
}

////////////// CG KERNEL 2: //////////////
__global__ void cuda_cg_2(int N,
                          int *csr_rowoffsets,
                          int *csr_colindices,
                          double *csr_values,
                          double *p,
                          double *Ap,
                          double *result1,
                          double *result2)
{

  __shared__ double shared_mem1[512];
  __shared__ double shared_mem2[512];

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
  {
    // line 5: get Ap
    double sum = 0;
    for (int k = csr_rowoffsets[i]; k < csr_rowoffsets[i + 1]; k++)
    {
      sum += csr_values[k] * p[csr_colindices[k]];
    }
    Ap[i] = sum;
  }

  // line 6: get dot(Ap,Ap) and dot(p,Ap)
  double dot1 = 0;
  double dot2 = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
  {
    dot1 += Ap[i] * Ap[i];
    dot2 += p[i] * Ap[i];
  }

  shared_mem1[threadIdx.x] = dot1;
  shared_mem2[threadIdx.x] = dot2;
  for (int k = blockDim.x / 2; k > 0; k /= 2)
  {
    __syncthreads();
    if (threadIdx.x < k)
    {
      shared_mem1[threadIdx.x] += shared_mem1[threadIdx.x + k];
      shared_mem2[threadIdx.x] += shared_mem2[threadIdx.x + k];
    }
  }

  if (threadIdx.x == 0)
  {
    atomicAdd(result1, shared_mem1[0]);
    atomicAdd(result2, shared_mem2[0]);
  }
}

/** Implementation of the conjugate gradient algorithm.
 *
 *  The control flow is handled by the CPU.
 *  Only the individual operations (vector updates, dot products, sparse
 * matrix-vector product) are transferred to CUDA kernels.
 *
 *  The temporary arrays p, r, and Ap need to be allocated on the GPU for use
 * with CUDA. Modify as you see fit.
 */
void conjugate_gradient(int N, // number of unknows
                        int *csr_rowoffsets, int *csr_colindices,
                        double *csr_values, double *rhs, double *solution)
//, double *init_guess)   // feel free to add a nonzero initial guess as needed
{
  // initialize timer
  Timer timer;

  // clear solution vector (it may contain garbage values):
  std::fill(solution, solution + N, 0);

  // initialize work vectors:
  double alpha, beta, residual_norm_squared, dot_pAp, dot_ApAp;
  double *cuda_solution, *cuda_p, *cuda_r, *cuda_Ap, *cuda_scalar, *cuda_dot_pAp, *cuda_dot_ApAp;
  hipMalloc(&cuda_p, sizeof(double) * N);
  hipMalloc(&cuda_r, sizeof(double) * N);
  hipMalloc(&cuda_Ap, sizeof(double) * N);
  hipMalloc(&cuda_solution, sizeof(double) * N);
  hipMalloc(&cuda_scalar, sizeof(double));

  hipMalloc(&cuda_dot_ApAp, sizeof(double));
  hipMalloc(&cuda_dot_pAp, sizeof(double));

  hipMemcpy(cuda_p, rhs, sizeof(double) * N, hipMemcpyHostToDevice);
  hipMemcpy(cuda_r, rhs, sizeof(double) * N, hipMemcpyHostToDevice);
  hipMemcpy(cuda_solution, solution, sizeof(double) * N, hipMemcpyHostToDevice);

  // get residual_norm_squared
  const double zero = 0;
  hipMemcpy(cuda_scalar, &zero, sizeof(double), hipMemcpyHostToDevice);
  cuda_dot_product<<<512, 512>>>(N, cuda_r, cuda_r, cuda_scalar);
  hipMemcpy(&residual_norm_squared, cuda_scalar, sizeof(double), hipMemcpyDeviceToHost);

  double initial_residual_squared = residual_norm_squared;

  // line 1: get alpha0, beta0, Ap0
  cuda_csr_matvec_product<<<512, 512>>>(N, csr_rowoffsets, csr_colindices, csr_values, cuda_p, cuda_Ap);

  hipMemcpy(cuda_scalar, &zero, sizeof(double), hipMemcpyHostToDevice);
  cuda_dot_product<<<512, 512>>>(N, cuda_p, cuda_Ap, cuda_scalar);
  hipMemcpy(&dot_pAp, cuda_scalar, sizeof(double), hipMemcpyDeviceToHost);
  alpha = residual_norm_squared / dot_pAp;

  hipMemcpy(cuda_scalar, &zero, sizeof(double), hipMemcpyHostToDevice);
  cuda_dot_product<<<512, 512>>>(N, cuda_Ap, cuda_Ap, cuda_scalar);
  hipMemcpy(&dot_ApAp, cuda_scalar, sizeof(double), hipMemcpyDeviceToHost);

  beta = (alpha * alpha * dot_ApAp - residual_norm_squared) / residual_norm_squared;

  

  int iters = 0;
  hipDeviceSynchronize();
  timer.reset();
  while (1)
  {

    hipMemcpy(cuda_scalar, &zero, sizeof(double), hipMemcpyHostToDevice);

    // std::cout << alpha << ", " << residual_norm_squared << std::endl;
    cuda_cg_1<<<512, 512>>>(N, alpha, beta, cuda_solution, cuda_r, cuda_p, cuda_Ap, cuda_scalar);

    hipMemcpy(&residual_norm_squared, cuda_scalar, sizeof(double), hipMemcpyDeviceToHost);

    // std::cout << residual_norm_squared << std::endl;

    hipMemcpy(cuda_dot_ApAp, &zero, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cuda_dot_pAp, &zero, sizeof(double), hipMemcpyHostToDevice);

    // std::cout << dot_ApAp << ", " << dot_pAp << ", " << residual_norm_squared << std::endl;

    cuda_cg_2<<<512, 512>>>(N, csr_rowoffsets, csr_colindices, csr_values,
                            cuda_p, cuda_Ap, cuda_dot_ApAp, cuda_dot_pAp);

    hipMemcpy(&dot_ApAp, cuda_dot_ApAp, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&dot_pAp, cuda_dot_pAp, sizeof(double), hipMemcpyDeviceToHost);
    // std::cout << alpha << ", " << residual_norm_squared << std::endl;

    // line 7:
    alpha = residual_norm_squared / dot_pAp;

    // line 8:
    beta = (alpha * alpha * dot_ApAp - residual_norm_squared) / residual_norm_squared;

    // std::cout << dot_ApAp << ", " << dot_pAp << std::endl;

    // check for convergence
    if (std::sqrt(residual_norm_squared / initial_residual_squared) < 1e-6)
    {
      break;
    }

    if (iters > 10000)
      break; // solver didn't converge
    ++iters;
  }
  hipMemcpy(solution, cuda_solution, sizeof(double) * N, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
  /*
  std::cout << "Time elapsed: " << timer.get() << " (" << timer.get() / iters << " per iteration)" << std::endl;

  if (iters > 10000)
    std::cout << "Conjugate Gradient did NOT converge within 10000 iterations"
              << std::endl;
  else
    std::cout << "Conjugate Gradient converged in " << iters << " iterations."
              << std::endl;
  */

  std::cout << timer.get() / iters << "," << std::endl;

  hipFree(cuda_p);
  hipFree(cuda_r);
  hipFree(cuda_Ap);
  hipFree(cuda_solution);
  hipFree(cuda_scalar);
}

/** Solve a system with `points_per_direction * points_per_direction` unknowns
 */
void solve_system(int points_per_direction)
{

  int N = points_per_direction *
          points_per_direction; // number of unknows to solve for

  // std::cout << "Solving Ax=b with " << N << " unknowns." << std::endl;

  //
  // Allocate CSR arrays.
  //
  // Note: Usually one does not know the number of nonzeros in the system matrix
  // a-priori.
  //       For this exercise, however, we know that there are at most 5 nonzeros
  //       per row in the system matrix, so we can allocate accordingly.
  //
  int *csr_rowoffsets = (int *)malloc(sizeof(double) * (N + 1));
  int *csr_colindices = (int *)malloc(sizeof(double) * 5 * N);
  double *csr_values = (double *)malloc(sizeof(double) * 5 * N);

  int *cuda_csr_rowoffsets, *cuda_csr_colindices;
  double *cuda_csr_values;
  //
  // fill CSR matrix with values
  //
  generate_fdm_laplace(points_per_direction, csr_rowoffsets, csr_colindices,
                       csr_values);

  //
  // Allocate solution vector and right hand side:
  //
  double *solution = (double *)malloc(sizeof(double) * N);
  double *rhs = (double *)malloc(sizeof(double) * N);
  std::fill(rhs, rhs + N, 1);

  //
  // Allocate CUDA-arrays //
  //
  hipMalloc(&cuda_csr_rowoffsets, sizeof(double) * (N + 1));
  hipMalloc(&cuda_csr_colindices, sizeof(double) * 5 * N);
  hipMalloc(&cuda_csr_values, sizeof(double) * 5 * N);
  hipMemcpy(cuda_csr_rowoffsets, csr_rowoffsets, sizeof(double) * (N + 1), hipMemcpyHostToDevice);
  hipMemcpy(cuda_csr_colindices, csr_colindices, sizeof(double) * 5 * N, hipMemcpyHostToDevice);
  hipMemcpy(cuda_csr_values, csr_values, sizeof(double) * 5 * N, hipMemcpyHostToDevice);

  //
  // Call Conjugate Gradient implementation with GPU arrays
  //
  conjugate_gradient(N, cuda_csr_rowoffsets, cuda_csr_colindices, cuda_csr_values, rhs, solution);

  //
  // Check for convergence:
  //
  /*
  double residual_norm = relative_residual(N, csr_rowoffsets, csr_colindices, csr_values, rhs, solution);
  std::cout << "Relative residual norm: " << residual_norm
            << " (should be smaller than 1e-6)" << std::endl;
  */
  hipFree(cuda_csr_rowoffsets);
  hipFree(cuda_csr_colindices);
  hipFree(cuda_csr_values);
  free(solution);
  free(rhs);
  free(csr_rowoffsets);
  free(csr_colindices);
  free(csr_values);
}

int main()
{
  std::vector<int> N_vec = {10, 25, 50, 75, 100, 250, 500, 750, 1000};
  for (const auto &N : N_vec)
  {
    solve_system(N); // solves a system with 100*100 unknowns
  }

  return EXIT_SUCCESS;
}
