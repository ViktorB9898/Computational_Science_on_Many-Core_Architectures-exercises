#include "hip/hip_runtime.h"
/**
 * 360.252 - Computational Science on Many-Core Architectures
 * WS 2022/23, TU Wien
 *
 * Simplistic simulator for a disease of very immediate concern (DIVOC). Inspired by COVID-19 simulations.
 *
 * DISCLAIMER: This simulator is for educational purposes only.
 * It may be arbitrarily inaccurate and should not be used for drawing any conclusions about any actual virus.
 */

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "timer.hpp"
#include <vector>
#include <algorithm>
#include <iostream>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#define YEAR 10

//
// Data container for simulation input
//

typedef struct
{

  int population_size; // Number of people to simulate

  //// Configuration
  int mask_threshold;      // Number of cases required for masks
  int lockdown_threshold;  // Number of cases required for lockdown
  int infection_delay;     // Number of days before an infected person can pass on the disease
  int infection_days;      // Number of days an infected person can pass on the disease
  int starting_infections; // Number of infected people at the start of the year
  int immunity_duration;   // Number of days a recovered person is immune

  // for each day:
  int *contacts_per_day;            // number of other persons met each day to whom the disease may be passed on
  double *transmission_probability; // how likely it is to pass on the infection to another person

} SimInput_t;

__host__ __device__ void init_input(SimInput_t *input)
{
  input->population_size = 8916845; // Austria's population in 2020 according to Statistik Austria

  input->mask_threshold = 5000;
  input->lockdown_threshold = 50000;
  input->infection_delay = 5; // 5 to 6 days incubation period (average) according to WHO
  input->infection_days = 3;  // assume three days of passing on the disease
  input->starting_infections = 10;
  input->immunity_duration = 180; // half a year of immunity

  input->contacts_per_day = (int *)malloc(sizeof(int) * YEAR);
  input->transmission_probability = (double *)malloc(sizeof(double) * YEAR);
  for (int day = 0; day < YEAR; ++day)
  {
    input->contacts_per_day[day] = 6;                                                // arbitrary assumption of six possible transmission contacts per person per day, all year
    input->transmission_probability[day] = 0.2 + 0.1 * cos((day / (double) YEAR) * 2 * M_PI); // higher transmission in winter, lower transmission during summer
  }
}

typedef struct
{
  // for each day:
  int *active_infections; // number of active infected on that day (including incubation period)
  int *lockdown;          // 0 if no lockdown on that day, 1 if lockdown

  // for each person:
  int *is_infected; // 0 if healty, 1 if currently infected
  int *infected_on; // day of infection. negative if not yet infected. January 1 is Day 0.

} SimOutput_t;

//
// Initializes the output data structure (values to zero, allocate arrays)
//
__host__ __device__ void init_output(SimOutput_t *output, int population_size)
{
  output->active_infections = (int *)malloc(sizeof(int) * YEAR);
  output->lockdown = (int *)malloc(sizeof(int) * YEAR);
  for (int day = 0; day < 10; ++day)
  {
    output->active_infections[day] = 0;
    output->lockdown[day] = 0;
  }

  output->is_infected = (int *)malloc(sizeof(int) * population_size);
  output->infected_on = (int *)malloc(sizeof(int) * population_size);

  for (int i = 0; i < population_size; ++i)
  {
    output->is_infected[i] = 0;
    output->infected_on[i] = 0;
  }
}

void run_simulation(const SimInput_t *input, SimOutput_t *output)
{
  //
  // Init data. For simplicity we set the first few people to 'infected'
  //
  for (int i = 0; i < input->population_size; ++i)
  {
    output->is_infected[i] = (i < input->starting_infections) ? 1 : 0;
    output->infected_on[i] = (i < input->starting_infections) ? 0 : -1;
  }

  //
  // Run simulation
  //
  for (int day = 0; day < YEAR; ++day) // loop over all days of the year
  {
    //
    // Step 1: determine number of infections and recoveries
    //
    int num_infected_current = 0;
    int num_recovered_current = 0;
    for (int i = 0; i < input->population_size; ++i)
    {

      if (output->is_infected[i] > 0)
      {
        if (output->infected_on[i] > day - input->infection_delay - input->infection_days && 
            output->infected_on[i] <= day - input->infection_delay) // currently infected and incubation period over
          num_infected_current += 1;
        else if (output->infected_on[i] < day - input->infection_delay - input->infection_days)
          num_recovered_current += 1;
      }
    }

    output->active_infections[day] = num_infected_current;
    if (num_infected_current > input->lockdown_threshold)
    {
      output->lockdown[day] = 1;
    }
    if (day > 0 && output->lockdown[day - 1] == 1)
    { // end lockdown if number of infections has reduced significantly
      output->lockdown[day] = (num_infected_current < input->lockdown_threshold / 3) ? 0 : 1;
    }
    char lockdown[] = " [LOCKDOWN]";
    char normal[] = "";
    printf("Day %d%s: %d active, %d recovered\n", day, output->lockdown[day] ? lockdown : normal, num_infected_current, num_recovered_current);

    //
    // Step 2: determine today's transmission probability and contacts based on pandemic situation
    //
    double contacts_today = input->contacts_per_day[day];
    double transmission_probability_today = input->transmission_probability[day];
    if (num_infected_current > input->mask_threshold)
    { // transmission is reduced with masks. Arbitrary factor: 2
      transmission_probability_today /= 2.0;
    }
    if (output->lockdown[day])
    { // contacts are significantly reduced in lockdown. Arbitrary factor: 4
      contacts_today /= 4;
    }

    //
    // Step 3: pass on infections within population
    //
    for (int i = 0; i < input->population_size; ++i) // loop over population
    {
      if (output->is_infected[i] > 0 && output->infected_on[i] > day - input->infection_delay - input->infection_days // currently infected
          && output->infected_on[i] <= day - input->infection_delay)                                                  // already infectious
      {
        // pass on infection to other persons with transmission probability
        for (int j = 0; j < contacts_today; ++j)
        {
          double r = ((double)rand()) / (double)RAND_MAX; // random number between 0 and 1
          if (r < transmission_probability_today)
          {
            r = ((double)rand()) / (double)RAND_MAX; // new random number to determine a random other person to transmit the virus to
            int other_person = r * input->population_size;
            if (output->is_infected[other_person] == 0                                 // other person is not infected
                || output->infected_on[other_person] < day - input->immunity_duration) // other person has no more immunity
            {
              output->is_infected[other_person] = 1;
              output->infected_on[other_person] = day;
            }
          }

        } // for contacts_per_day
      }   // if currently infected
    }     // for i

  } // for day
}


// ########## TASK b) ##########

// linear congruential generator (poor rng but should be enough)
__global__ void LCG(double *random_vec, int N, int seed)
{
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  // individual starting value for each thread
  int s = seed + id;
  double tmp;
  for (int i = id; i < N; i += blockDim.x * gridDim.x)
  {
    s = s * 1234567891 + 54321; // some random parameters
    tmp = (s % 100);
    //printf("%g\n", tmp);
    random_vec[i] = tmp / 200 + 0.5f; // s.t. we get numbers between 0 and 1
    printf("%g\n", random_vec[i]);
  }
}

// ########## TASK c) ##########

__global__ void initialize(SimInput_t *input, SimOutput_t *output)
{
  init_input(input);
  printf("TEST: mask_threshold %d\n", input->mask_threshold);
  init_output(output, (int) &input->population_size);
}

// test if input has been initialized correctly
__global__ void test_init(SimInput_t *input, SimOutput_t *output)
{
  printf("TEST: mask_threshold %d\n", input->mask_threshold);
}

//__global__
void run(SimInput_t *input, SimOutput_t *output, double *random_vec)
{
  printf("\nGPU RUN STARTS HERE\n");

  int rng_count = 0;
  //
  // Init data. For simplicity we set the first few people to 'infected'
  //
  for (int i = 0; i < input->population_size; ++i)
  {
    output->is_infected[i] = (i < input->starting_infections) ? 1 : 0;
    output->infected_on[i] = (i < input->starting_infections) ? 0 : -1;
  }

  //
  // Run simulation
  //
  for (int day = 0; day < YEAR; ++day) // loop over all days of the year
  {
    //
    // Step 1: determine number of infections and recoveries
    //
    int num_infected_current = 0;
    int num_recovered_current = 0;
    for (int i = 0; i < input->population_size; ++i)
    {

      if (output->is_infected[i] > 0)
      {
        if (output->infected_on[i] > day - input->infection_delay - input->infection_days && 
            output->infected_on[i] <= day - input->infection_delay) // currently infected and incubation period over
          num_infected_current += 1;
        else if (output->infected_on[i] < day - input->infection_delay - input->infection_days)
          num_recovered_current += 1;
      }
    }

    output->active_infections[day] = num_infected_current;
    if (num_infected_current > input->lockdown_threshold)
    {
      output->lockdown[day] = 1;
    }
    if (day > 0 && output->lockdown[day - 1] == 1)
    { // end lockdown if number of infections has reduced significantly
      output->lockdown[day] = (num_infected_current < input->lockdown_threshold / 3) ? 0 : 1;
    }
    char lockdown[] = " [LOCKDOWN]";
    char normal[] = "";
    printf("Day %d%s: %d active, %d recovered\n", day, output->lockdown[day] ? lockdown : normal, num_infected_current, num_recovered_current);
    printf("Random numbers so far: %d, Test rng number: %g\n", rng_count, random_vec[rng_count]);
    //
    // Step 2: determine today's transmission probability and contacts based on pandemic situation
    //
    double contacts_today = input->contacts_per_day[day];
    double transmission_probability_today = input->transmission_probability[day];
    if (num_infected_current > input->mask_threshold)
    { // transmission is reduced with masks. Arbitrary factor: 2
      transmission_probability_today /= 2.0;
    }
    if (output->lockdown[day])
    { // contacts are significantly reduced in lockdown. Arbitrary factor: 4
      contacts_today /= 4;
    }

    //
    // Step 3: pass on infections within population
    //
    for (int i = 0; i < input->population_size; ++i) // loop over population
    {
      if (output->is_infected[i] > 0 && output->infected_on[i] > day - input->infection_delay - input->infection_days // currently infected
          && output->infected_on[i] <= day - input->infection_delay)                                                  // already infectious
      {
        // pass on infection to other persons with transmission probability
        for (int j = 0; j < contacts_today; ++j)
        {
          double r = random_vec[rng_count++]; // random number between 0 and 1
          if (r < transmission_probability_today)
          {
            r = random_vec[rng_count++]; // new random number to determine a random other person to transmit the virus to
            int other_person = r * input->population_size;
            if (output->is_infected[other_person] == 0                                 // other person is not infected
                || output->infected_on[other_person] < day - input->immunity_duration) // other person has no more immunity
            {
              output->is_infected[other_person] = 1;
              output->infected_on[other_person] = day;
            }
          }

        } // for contacts_per_day
      }   // if currently infected
    }     // for i

  } // for day
}

// for a)
float random_function(){
  return (float) rand()/RAND_MAX;
}

int main(int argc, char **argv)
{
  int N_BLOCKS = 256;
  int N_THREADS = 256;

  // temporary
  int N = 10;

  // ########## TASK a) ##########
  // create random points and copy them over to GPU
  double *random_vec = (double *)malloc(sizeof(double) * N);
  std::generate(random_vec, random_vec + N, random_function);

  double *cuda_random_vec;
  hipMalloc(&cuda_random_vec, sizeof(double) * N);
  hipMemcpy(cuda_random_vec, random_vec, sizeof(double) * N, hipMemcpyHostToDevice);

  // ########## TASK b) ##########
  // random_vec will be overwritten by this function
  LCG<<<N_BLOCKS, N_THREADS>>>(cuda_random_vec, N, 42);

  // ########## TASK c) ##########
  // create input and output strucs on CPU and copy them over to GPU
  SimInput_t input;
  SimOutput_t output;
  SimInput_t *cuda_input;
  hipMalloc(&cuda_input, sizeof(SimInput_t));
  SimOutput_t *cuda_output;
  hipMalloc(&cuda_output, sizeof(SimOutput_t));

  hipMemcpy(cuda_input, &input, sizeof(SimInput_t), hipMemcpyHostToDevice);
  hipMemcpy(cuda_output, &output, sizeof(SimOutput_t), hipMemcpyHostToDevice);

  // let a single thread do the initialization
  initialize<<<1, 1>>>(cuda_input, cuda_output);

  hipDeviceSynchronize();


  // test if initialize function worked
  test_init<<<1,1>>>(cuda_input, cuda_output);

  hipDeviceSynchronize();

  // hipMemcpy(&input, cuda_input, sizeof(SimInput_t), hipMemcpyDeviceToHost);
  

  init_input(&input);
  init_output(&output, input.population_size);

  Timer timer;
  srand(0); // initialize random seed for deterministic output
  timer.reset();
  run_simulation(&input, &output);
  printf("Simulation time: %g\n", timer.get());

  hipDeviceSynchronize();

  //init_input(&input);
  //init_output(&output, input.population_size);

  // tried to make it work at first on CPU but segmentation fault ...
  //run(&input, &output, random_vec);

  // next step would have been to make it run with one thread on GPU
  //run<<<1,1>>>(cuda_input, cuda_output, cuda_random_vec);



  return EXIT_SUCCESS;
}